#include <hip/hip_runtime.h>
#include <iostream>

__global__ void addKernel(int *a, int *b, int *c, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        c[idx] = a[idx] + b[idx];
    }
}

int main() {
    const int arraySize = 5;
    const int a[arraySize] = {1, 2, 3, 4, 5};
    const int b[arraySize] = {10, 20, 30, 40, 50};
    int c[arraySize] = {0};

    int *dev_a = nullptr;
    int *dev_b = nullptr;
    int *dev_c = nullptr;

    hipMalloc((void**)&dev_a, arraySize * sizeof(int));
    hipMalloc((void**)&dev_b, arraySize * sizeof(int));
    hipMalloc((void**)&dev_c, arraySize * sizeof(int));

    hipMemcpy(dev_a, a, arraySize * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, arraySize * sizeof(int), hipMemcpyHostToDevice);

    addKernel<<<1, arraySize>>>(dev_a, dev_b, dev_c, arraySize);

    hipMemcpy(c, dev_c, arraySize * sizeof(int), hipMemcpyDeviceToHost);

    std::cout << "Result: ";
    for (int i = 0; i < arraySize; i++) {
        std::cout << c[i] << " ";
    }
    std::cout << std::endl;

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    return 0;
}